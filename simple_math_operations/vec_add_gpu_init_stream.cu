#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>


// On GPU section
__global__
void initWithNumber(float num, float *vec, int N)
{

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride) {
        vec[i] = num;
    }
}

__global__
void addVectors(float *result, const float *a, const float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride) {
        result[i] = a[i] + b[i];
    }
}

// On CPU section
void verifyResult(float target, float *vector, int N)
{
    for(int i = 0; i < N; i++) {
        if(vector[i] != target) {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main()
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    const int N = 2<<24;
    size_t size = N * sizeof(float);

    float *a, *b;
    float *c;

    hipError_t addVectorsErr;
    hipError_t asyncErr;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // inform GPU it will need those (works only for Linux)
//    hipMemPrefetchAsync(a, size, deviceId);
//    hipMemPrefetchAsync(b, size, deviceId);
//    hipMemPrefetchAsync(c, size, deviceId);

    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error when prefetch: %s\n", hipGetErrorString(addVectorsErr));

    size_t threadsPerBlock = 1024;
    size_t numberOfBlocks = 32 * numberOfSMs;

    hipStream_t streamA;
    hipStream_t streamB;
    hipStream_t streamC;

    hipStreamCreate(&streamA);
    hipStreamCreate(&streamB);
    hipStreamCreate(&streamC);

    initWithNumber<<<numberOfBlocks, threadsPerBlock, 0, streamA>>>(3, a, N);
    initWithNumber<<<numberOfBlocks, threadsPerBlock, 0, streamB>>>(4, b, N);
    initWithNumber<<<numberOfBlocks, threadsPerBlock, 0, streamC>>>(0, c, N);

    addVectors<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    verifyResult(7, c, N);

    // Do not forget to destroy streams
    hipStreamDestroy(streamA);
    hipStreamDestroy(streamB);
    hipStreamDestroy(streamC);

    // and free memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
}
